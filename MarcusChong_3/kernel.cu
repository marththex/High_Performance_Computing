
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <iostream> 
#define N 512
__global__ void kernel(int *a[], int n) 
{
	int index = threadIdx.x + blockDim.x * threadIdx.y;
	if(index < n)
	a[index] = &index;
	
}

int main(int argc, char **argv) 
{
	int n = N;
	if(argc == 2)
		n = atoi(argv[1]);
	else if (argc == 1)
		n = 512;
	else
		printf("Error # of Arguments");
	
	int* a = new int[n];
	int *dev_a[N];
	
	//allocate the memory on the GPU
	hipMalloc( (void**)&dev_a, N * sizeof(int) );
	
	//creates a kernel
	kernel<<<1,N>>>(dev_a, n); 
	
	//copy the arrays 'a' to the GPU
	hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );

	for (int i=0; i< n; i++) 
	{
		printf("%d\n",a[i]);
	}

	hipFree( dev_a );
	delete[] a;
	
    return 0;
}
